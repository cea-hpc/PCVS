#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA
// with an array of offsets. Then the offsets are added in parallel
// to produce the string "World!"
// By Ingemar Ragnemalm 2010

// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <assert.h>
#define safe_cudart(u)                                                         \
	if (u != hipSuccess) {                                                    \
		fprintf(stderr, "Error %d: %s\n", __LINE__, hipGetErrorString(u));   \
		abort();                                                               \
	}

const int N = 16; 
const int blocksize = 16; 

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}

int rank = -1;

int main(int argc, char ** argv)
{
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	char a[N] = "Hello \0\0\0\0\0\0";
	char final[N];
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	sprintf(final, "%s", a);

	safe_cudart(hipMalloc( (void**)&ad, csize ));
	safe_cudart(hipMalloc( (void**)&bd, isize )); 
	safe_cudart(hipMemcpy( ad, a, csize, hipMemcpyHostToDevice )); 
	safe_cudart(hipMemcpy( bd, b, isize, hipMemcpyHostToDevice )); 
	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid( 10, 1, 1 );

	hello<<<dimGrid, dimBlock>>>(ad, bd);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
	{
		printf("KERNEL FAILED: %s\n", hipGetErrorString(hipGetLastError()));
		abort();
	}

	safe_cudart(hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost )); 
	safe_cudart(hipFree( ad ));
	safe_cudart(hipFree( bd ));

	sprintf(final, "%s%s", final, a);
	fprintf(stderr, "Rank %2d: %s\n", rank, final);
	assert(strcmp(final, "Hello World!") == 0);
	MPI_Finalize();
	return EXIT_SUCCESS;
}
