#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define safe_cudart(u)                                                         \
	if (u != hipSuccess) {                                                    \
		fprintf(stderr, "Error %d: %s\n", __LINE__, hipGetErrorString(u));   \
		abort();                                                               \
	}

int rank = -1;

	__global__
void square(int* in, int* out)
{
	(*out) = (*in) * (*in);
}

int main(int argc, char **argv)
{
	int comm_size;

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
	
	int *in, *out, *gin, *gout, check = 0, reduce = 0;

	in  = (int*) malloc(sizeof(int));
	out = (int*) malloc(sizeof(int));

	safe_cudart(hipMalloc((void**)&gin, sizeof(int)));
	safe_cudart(hipMalloc((void**)&gout, sizeof(int)));
	
	*in = rank;
	*out = -1;

	safe_cudart(hipMemcpy(gin, in, sizeof(int), hipMemcpyHostToDevice));

	/* compute the square of the current rank */
	square<<<1, 1>>>(gin, gout);

	safe_cudart(hipMemcpy(out, gout, sizeof(int), hipMemcpyDeviceToHost));

	safe_cudart(hipFree(gin));
	safe_cudart(hipFree(gout));

	/* sum all rank squares */
	MPI_Reduce(out, &reduce, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);

	MPI_Finalize();
	
	int i;
	for (i = 0; i < comm_size; ++i)
	{
		check += (i * i);
	}

	if(!rank)
	{
		if(check != reduce)
		{
			fprintf(stderr, "Error: Expected %d and get %d\n", check, reduce);
			return 1;
		}
		else
		{
			fprintf(stderr, "Test Successfull !\n");
		}
	}
	return 0;
}
